#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define rms_alpha 0.9

static __global__ void kerd_opti_rmsprop(
	float * p, float * dp, float * g,
	float alpha, uint POIDS, float div)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		float _grad = dp[thx] / div;
		float _g = rms_alpha*g[thx] + _grad*_grad;
		p[thx] -= alpha * _grad / (sqrtf(_g) + 1e-5);
		g[thx] = _g;
	}
};

static __global__ void kerd_opti_rmsprop_masque(
	float * p, float * dp, float * g,
	float alpha, uint POIDS, float div, uint * masque)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		if (masque[thx] == NON_MASQUEE) {
			float _grad = dp[thx] / div;
			float _g = rms_alpha*g[thx] + _grad*_grad;
			p[thx] -= alpha * _grad / (sqrtf(_g) + 1e-5);
			g[thx] = _g;
		}
	}
};

Rmsprop_t * cree_rmsprop(
	Mdl_t * mdl)
{
	Rmsprop_t * ret = alloc<Rmsprop_t>(1);
	ret->g[0] = cudazero<float>(mdl->Y[0]*N);
	FOR(1, c, C) ret->g[c] = cudazero<float>(mdl->inst_POIDS[c]);
	return ret;
};

void liberer_rmsprop(Rmsprop_t * rmsprop) {
	FOR(0, c, C) cudafree<float>(rmsprop->g[c]);
	free(rmsprop);
};

void opti_rmsprop(
	Mdl_t * mdl, Rmsprop_t * rmsprop,
	float * alpha, float div, uint ** masque)
{
	//	Filtres
	uint FILTRES = mdl->Y[0];	//pas de *N, car c'est le filtre qu'on ignore, pas les points
	if (masque == 0) {
		kerd_opti_rmsprop<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->p__d[0], mdl->dp__d[0], rmsprop->g[0], alpha[0], FILTRES, div);
	} else {
		kerd_opti_rmsprop_masque<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->p__d[0], mdl->dp__d[0], rmsprop->g[0], alpha[0], FILTRES, div, masque[0]
		);
	}
	//	Poids
	FOR(1, c, C) {
		uint POIDS = mdl->inst_POIDS[c];
		
		if (masque == 0) {
			kerd_opti_rmsprop<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], rmsprop->g[c],
				alpha[c], POIDS, div
			);
		} else {
			kerd_opti_rmsprop_masque<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], rmsprop->g[c],
				alpha[c], POIDS, div, masque[c]
			);
		}
	};
	ATTENDRE_CUDA();
	mdl_gpu_vers_cpu(mdl);
};