#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static __global__ void kerd_opti_simple(
	float * p, float * dp, float alpha, uint POIDS, float div)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		p[thx] -= alpha * dp[thx] / div;
	}
};

static __global__ void kerd_opti_simple_masque(
	float * p, float * dp, float alpha, uint POIDS, float div, uint * masque)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < POIDS) {
		if (masque[thx] == NON_MASQUEE)
			p[thx] -= alpha * dp[thx] / div;
	}
};

void opti_simple(Mdl_t * mdl, float * alpha, float div, uint ** masque) {
	//	Filtres
	uint FILTRES = mdl->Y[0];	//pas de *N, car c'est le filtre qu'on ignore, pas les points
	if (masque == 0) {
		kerd_opti_simple<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->p__d[0], mdl->dp__d[0], alpha[0], FILTRES, div);
	} else {
		kerd_opti_simple_masque<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->p__d[0], mdl->dp__d[0], alpha[0], FILTRES, div, masque[0]
		);
	}
	//	Poids
	FOR(1, c, C) {
		uint POIDS = mdl->inst_POIDS[c];
		if (masque == 0) {
			kerd_opti_simple<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], alpha[c], POIDS, div
			);
		} else {
			kerd_opti_simple_masque<<<dim3(KERD(POIDS, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], alpha[c], POIDS, div, masque[c]
			);
		}
	};
	ATTENDRE_CUDA();
	mdl_gpu_vers_cpu(mdl);
};