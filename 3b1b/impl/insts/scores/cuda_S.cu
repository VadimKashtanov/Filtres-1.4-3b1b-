#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../../impl_tmpl/tmpl_etc.cu"

//	===============================================================

static __global__ void kerd_nvidia_score_somme(
	float * y, uint depart, uint T,
	float * score, float * _PRIXS)
{
	float s = 0;
	FOR(0, i, T) {
		FOR(0, p, P) {
			s += (P-p)*cuda_SCORE(
				y[(depart+i)*P+p], _PRIXS[depart+i+p+1], _PRIXS[depart+i/*+p*/]
			);
		}
	}
	*score = s / (float)(T*P);
};

float nvidia_score(float * y, uint depart, uint T)
{
	float * score__d = cudalloc<float>(1);
	kerd_nvidia_score_somme<<<1,1>>>(
		y, depart, T,
		score__d, prixs__d
	);
	ATTENDRE_CUDA();
	float _score;
	CONTROLE_CUDA(hipMemcpy(&_score, score__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipFree(score__d));
	return _score;
};

//	===============================================================

static __global__ void kerd_nvidia_prediction_somme(
	float * y, uint depart, uint T,
	float * pred, float * _PRIXS,
	uint canal_p)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	if (thx < T) {
		float p1 = _PRIXS[depart+thx+canal_p+1];
		float p0 = _PRIXS[depart+thx/*+canal_p*/];
		atomicAdd(
			pred,
			1.0*(uint)(cuda_signe((y[(depart+thx)*P+canal_p])) == cuda_signe((p1/p0-1)))
		);
	};
};

static float __nvidia_prediction(float * y, uint depart, uint T, uint canal_p) {
	float * pred__d = cudalloc<float>(1);
	CONTROLE_CUDA(hipMemset(pred__d, 0, 1*sizeof(float)));
	kerd_nvidia_prediction_somme<<<dim3(KERD(T,1024)),dim3(1024)>>>(
		y, depart, T,
		pred__d, prixs__d,
		canal_p
	);
	ATTENDRE_CUDA();
	float _pred;
	CONTROLE_CUDA(hipMemcpy(&_pred, pred__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	cudafree<float>(pred__d);
	return _pred / (float)T;
};

float * nvidia_prediction(float * y, uint depart, uint T) {
	float * pred = (float*)malloc(sizeof(float) * P);
	FOR(0, p, P) pred[p] = __nvidia_prediction(y, depart, T, p);
	return pred;
};
//	===============================================================

static __global__ void kerd_nvidia_score_dpowf(
	float * y, float * dy,
	uint depart, uint T,
	float * _PRIXS)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;

	if (_t < T) {
		FOR(0, p, P) {
			dy[(depart+_t)*P+p] = (P-p)*cuda_dSCORE(
				y[(depart+_t)*P+p], _PRIXS[depart+_t+p+1], _PRIXS[depart+_t/*+p*/]
			) / ((float)T*P);
		}
	}
};

void d_nvidia_score(float * y, float * dy, uint depart, uint T) {
	kerd_nvidia_score_dpowf<<<dim3(KERD(T,1024)), dim3(1024)>>>(
		y, dy,
		depart, T,
		prixs__d
	);
	ATTENDRE_CUDA();
};