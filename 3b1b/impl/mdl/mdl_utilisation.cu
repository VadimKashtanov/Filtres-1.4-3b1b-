#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

float mdl_score(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode);
	if (mode == 0) return  intel_score(mdl->y[C-1], t0, (t1-t0));
	else           return nvidia_score(mdl->y__d[C-1], t0, (t1-t0));
};

float* mdl_pred(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode);
	if (mode == 0) return  intel_prediction(mdl->y[C-1], t0, (t1-t0));
	else           return nvidia_prediction(mdl->y__d[C-1], t0, (t1-t0));
};

float mdl_gain(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode);
	if (mode != 0)
		CONTROLE_CUDA(hipMemcpy(mdl->y[C-1], mdl->y__d[C-1], sizeof(float)*PRIXS, hipMemcpyDeviceToHost))
	//
	float u = 100;
	FOR(t0, t, t1) {
		u += u*25*mdl->y[C-1][t*P+0]*(prixs[t+1]/prixs[t]-1);
		if (u < 0) u = 0;
	};
	return u - 100;
};

void mdl_aller_retour(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	mdl_f(mdl, t0, t1, mode);
	if (mode == 0) mdl_zero_deriv_cpu(mdl);
	else           mdl_zero_deriv_gpu(mdl);
	if (mode == 0) d_intel_score(mdl->y[C-1], mdl->dy[C-1], t0, (t1-t0));
	else           d_nvidia_score(mdl->y__d[C-1], mdl->dy__d[C-1], t0, (t1-t0));
	mdl_df(mdl, t0, t1, mode);
};